/**
 * NormmixpdfSlowGPU.cu
 * Author:       Vilius Narbutas
 * Date:         2013
 * Description:  This program implements GPU version of 'for' loop in 
 *          normmixpdf_slow. 
 * Compilation:  nvcc -arch=sm_20 -ptx NormmixpdfSlowGPU.cu
 * Requirements: Graphics card that supports >2x architecture.
 *
 * Notes:        These functions support only 1D data arrays at the moment.
 **
 */

//#define _USE_MATH_DEFINES

#include <hip/hip_runtime.h>
#include <cmath>

// This program implements GPU version of normmixpdf_slow.
__global__ void NormmixpdfSlowGPU(double *p, double const *cov, 
        double const *mu, double const *w, double const *x, 
        double const dn, double const a, double const size_x)
{
    int id_x = threadIdx.x;
    int id_y = blockIdx.y;
    
    for (int i = id_x*1000+id_y; i < size_x; i+=1000)
    {
        double dx = 0;
        dx = x[i] - mu[id_x];
        p[i] = w[id_x] * (1/(a*sqrt(cov[id_x])))*exp (-0.5*dx*dx/(cov[id_x]+dn));
    }
}